#include <iostream>
#include "KittenLBVH/lbvh.cuh"

#include <unordered_set>

void main(int arg, char** args)
{
    // Tests the LBVH with a simple test case of 100k objects.

    const int   N = 100000;
    const float R = 0.001f;

    printf("Generating Data...\n");
    std::vector<Kitten::Bound<3, float>> points(N);

    srand(1);
    for (size_t i = 0; i < N; i++) {
        Kitten::Bound<3, float> b(Kitten::vec3(rand() / (float)RAND_MAX,
                                               rand() / (float)RAND_MAX,
                                               rand() / (float)RAND_MAX));
        b.pad(R);
        points[i] = b;
    }

    thrust::device_vector<Kitten::Bound<3, float>> d_points(points.begin(),
                                                            points.end());

    thrust::device_vector<Kitten::ivec2> d_res(100 * N);
    hipDeviceSynchronize();

    // Build BVH
    Kitten::LBVH bvh;
    printf("Building LBVH...\n");
    bvh.compute(thrust::raw_pointer_cast(d_points.data()), N);
    hipDeviceSynchronize();

    // Query BVH
    printf("Querying LBVH...\n");
    int numCols =
        bvh.query(thrust::raw_pointer_cast(d_res.data()), d_res.size());

    // Print results
    printf("Getting results...\n");
    thrust::host_vector<Kitten::ivec2> res(d_res.begin(),
                                           d_res.begin() + numCols);

    printf("%d collision pairs found on GPU.\n", res.size());
    // printf("GPU:\n");
    // for (size_t i = 0; i < res.size(); i++)
    // 	printf("%d %d\n", res[i].x, res[i].y);

    // Brute force compute the same result
    std::unordered_set<Kitten::ivec2> resSet;
    bool                              good = true;

    for (size_t i = 0; i < res.size(); i++) {
        Kitten::ivec2 a = res[i];
        if (a.x > a.y)
            std::swap(a.x, a.y);
        if (!resSet.insert(a).second) {
            printf("Error: Duplicate result\n");
            good = false;
        }
    }

    int numCPUFound = 0;
    printf("\nRunning brute force CPU collision detection...\n");
    for (int i = 0; i < N; i++)
        for (int j = i + 1; j < N; j++)
            if (points[i].intersects(points[j])) {
                numCPUFound++;
                if (resSet.find(Kitten::ivec2(i, j)) == resSet.end()) {
                    printf(
                        "Error: CPU result %d %d not found in GPU "
                        "result.\n",
                        i,
                        j);
                    good = false;
                }
            }

    if (numCPUFound != res.size()) {
        printf("Error: CPU and GPU results do not match\n");
        good = false;
    }

    printf("%d collision pairs found on CPU.\n", numCPUFound);
    printf(good ? "CPU and GPU results match.\n" :
                  "CPU and GPU results MISMATCH!\n");

    bvh.bvhSelfCheck();
}