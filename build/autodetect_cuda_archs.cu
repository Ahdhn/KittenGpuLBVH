		
#include <hip/hip_runtime.h>
#include <stdio.h>
		int main() {
		int count = 0; 
		if (hipSuccess != hipGetDeviceCount(&count)) { return -1; }
		if (count == 0) { return -1; }
		for (int device = 0; device < count; ++device) {
			hipDeviceProp_t prop; 
			bool is_unique = true; 
			if (hipSuccess == hipGetDeviceProperties(&prop, device)) {
				for (int device_1 = device - 1; device_1 >= 0; --device_1) {
					hipDeviceProp_t prop_1; 
					if (hipSuccess == hipGetDeviceProperties(&prop_1, device_1)) {
						if (prop.major == prop_1.major && prop.minor == prop_1.minor) {
							is_unique = false; 
							break; 
						}
					}
					else { return -1; }
				}
				if (is_unique) {
					fprintf(stderr, "%d%d", prop.major, prop.minor);
				}
			}
			else { return -1; }
		}
		return 0; 
		}
		